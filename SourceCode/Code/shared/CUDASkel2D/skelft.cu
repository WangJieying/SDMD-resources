#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include "include/skelft.h"
#include <stdio.h>



// Parameters for CUDA kernel executions; more or less optimized for a 1024x1024 image.
#define BLOCKX		16
#define BLOCKY		16
#define BLOCKSIZE	64
#define TILE_DIM	32
#define BLOCK_ROWS	16



/****** Global Variables *******/
const int NB = 7;						// Nr buffers we use and store in the entire framework
short2 **pbaTextures;					// Work buffers used to compute and store resident images
//	0: work buffer
//	1: FT
//	2: thresholded DT
//	3: thresholded skeleton
//	4: topology analysis
//  5: work buffer for topology
//  6: skeleton FT
//

float*			pbaTexSiteParam;		// Stores boundary parameterization
int				pbaTexSize;				// Texture size (squared) actually used in all computations
int				floodBand  = 4,			// Various FT computation parameters; defaults are good for an 1024x1024 image.	
				maurerBand = 4,
				colorBand  = 4;		

texture<short2> pbaTexColor;			// 2D textures (bound to various buffers defined above as needed)
texture<short2> pbaTexColor2;			//
texture<short2> pbaTexLinks;
texture<float>  pbaTexParam;			// 1D site parameterization texture (bound to pbaTexSiteParam)
texture<unsigned char>
				pbaTexGray;				// 2D texture of unsigned char values, e.g. the binary skeleton

#if __CUDA_ARCH__ < 110					// We cannot use atomic intrinsics on SM10 or below. Thus, we define these as nop.
#define atomicInc(a,b) 0				// The default will be that some code e.g. endpoint detection will thus not do anything.
#endif



/********* Kernels ********/
#include "include/skelftKernel.h"



// Initialize necessary memory (CPU/GPU sides)
// - textureSize: The max size of any image we will process until re-initialization
void skelft2DInitialization(int maxTexSize)
{
	hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,0);													// Query device properties, list something about them

    int pbaMemSize = maxTexSize * maxTexSize * sizeof(short2);								// A buffer has 2 shorts / pixel

    pbaTextures  = (short2 **) malloc(NB * sizeof(short2*));								// We will use NB buffers

	for(int i=0;i<NB;++i)
       hipMalloc((void **) &pbaTextures[i], pbaMemSize);									// Allocate work buffer 'i'

    hipMalloc((void **) &pbaTexSiteParam, maxTexSize * maxTexSize * sizeof(float));		// Sites texture 
}




// Deallocate all allocated memory
void skelft2DDeinitialization()
{
    for(int i=0;i<NB;++i) hipFree(pbaTextures[i]); 
	hipFree(pbaTexSiteParam);
    free(pbaTextures);
}



__global__ void kernelSiteParamInit(short2* inputVoro, int size)							//Initialize the Voronoi textures from the sites' encoding texture (parameterization)
{																							//REMARK: we interpret 'inputVoro' as a 2D texture, as it's much easier/faster like this
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx<size && ty<size)																	//Careful not to go outside the image..
	{
	  int i = TOID(tx,ty,size);
	  float param = tex1Dfetch(pbaTexParam,i);												//The sites-param has non-zero (parameter) values precisely on non-boundary points

	  short2& v = inputVoro[i];
	  v.x = v.y = MARKER;																	//Non-boundary points are marked as 0 in the parameterization. Here we will compute the FT.
	  if (param)																			//These are points which define the 'sites' to compute the FT/skeleton (thus, have FT==identity)
	  {																						//We could use an if-then-else here, but it's faster with an if-then
	     v.x = tx; v.y = ty;
	  }
	}
}



void skelft2DInitializeInput(float* sites, int size)										// Copy input sites from CPU to GPU; Also set up site param initialization in pbaTextures[0]
{
    pbaTexSize = size;																		// Size of the actual texture being used in this run; can be smaller than the max-tex-size
																							// which was used in skelft2DInitialization()

	hipMemcpy(pbaTexSiteParam, sites, pbaTexSize * pbaTexSize * sizeof(float), hipMemcpyHostToDevice);
																							// Pass sites parameterization to CUDA.  Must be done before calling the initialization
																							// kernel, since we use the sites-param as a texture in that kernel
	hipBindTexture(0, pbaTexParam, pbaTexSiteParam);										// Bind the sites-param as a 1D texture so we can quickly index it next
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);
	
	kernelSiteParamInit<<<grid,block>>>(pbaTextures[0],pbaTexSize);							// Do the site param initialization. This sets up pbaTextures[0]
	hipUnbindTexture(pbaTexParam);
}





// In-place transpose a squared texture. 
// Block orders are modified to optimize memory access. 
// Point coordinates are also swapped. 
void pba2DTranspose(short2 *texture)
{
    dim3 block(TILE_DIM, BLOCK_ROWS); 
    dim3 grid(pbaTexSize / TILE_DIM, pbaTexSize / TILE_DIM); 

    hipBindTexture(0, pbaTexColor, texture); 
    kernelTranspose<<< grid, block >>>(texture, pbaTexSize); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 1 of PBA. m1 must divides texture size
void pba2DPhase1(int m1, short xm, short ym, short xM, short yM) 
{
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexSize / block.x, m1); 

    // Flood vertically in their own bands
    hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
    kernelFloodDown<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1); 
    hipUnbindTexture(pbaTexColor); 

    hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelFloodUp<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1); 

    // Passing information between bands
    grid = dim3(pbaTexSize / block.x, m1); 
    kernelPropagateInterband<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m1); 

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelUpdateVertical<<< grid, block >>>(pbaTextures[1], pbaTexSize, m1, pbaTexSize / m1); 
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 2 of PBA. m2 must divides texture size
void pba2DPhase2(int m2) 
{
    // Compute proximate points locally in each band
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexSize / block.x, m2); 
    hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelProximatePoints<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelCreateForwardPointers<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    // Repeatly merging two bands into one
    for (int noBand = m2; noBand > 1; noBand /= 2) {
        grid = dim3(pbaTexSize / block.x, noBand / 2); 
        kernelMergeBands<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / noBand); 
    }

    // Replace the forward link with the X coordinate of the seed to remove
    // the need of looking at the other texture. We need it for coloring.
    grid = dim3(pbaTexSize / block.x, pbaTexSize); 
    kernelDoubleToSingleList<<< grid, block >>>(pbaTextures[0], pbaTexSize); 
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 3 of PBA. m3 must divides texture size
void pba2DPhase3(int m3) 
{
    dim3 block = dim3(BLOCKSIZE / m3, m3); 
    dim3 grid = dim3(pbaTexSize / block.x); 
    hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
    kernelColor<<< grid, block >>>(pbaTextures[1], pbaTexSize); 
    hipUnbindTexture(pbaTexColor); 
}



void skel2DFTCompute(short xm, short ym, short xM, short yM, int floodBand, int maurerBand, int colorBand)
{
    pba2DPhase1(floodBand,xm,ym,xM,yM);										//Vertical sweep

    pba2DTranspose(pbaTextures[1]);											//

    pba2DPhase2(maurerBand);												//Horizontal coloring

    pba2DPhase3(colorBand);													//Row coloring

    pba2DTranspose(pbaTextures[1]); 
}





__global__ void kernelThresholdDT(unsigned char* output, int size, float threshold2, short xm, short ym, short xM, short yM)
//Input:    pbaTexColor: closest-site-ids per pixel, i.e. FT
//Output:   output: thresholded DT
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (tx>xm && ty>ym && tx<xM && ty<yM)									//careful not to index outside the image..
	{	
  	  int    id     = TOID(tx, ty, size);
	  short2 voroid = tex1Dfetch(pbaTexColor,id);							//get the closest-site to tx,ty into voroid.x,.y
	  float  d2     = (tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y);
	  output[id]    = (d2<=threshold2);										//threshold DT into binary image
    }
}



__global__ void kernelDT(short* output, int size, float threshold2, short xm, short ym, short xM, short yM)
//Input:    pbaTexColor: closest-site-ids per pixel, i.e. FT
//Output:   output: DT
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
		
	if (tx>xm && ty>ym && tx<xM && ty<yM)									//careful not to index outside the image..
	{	
  	  int    id     = TOID(tx, ty, size);
	  short2 voroid = tex1Dfetch(pbaTexColor,id);							//get the closest-site to tx,ty into voroid.x,.y
	  float  d2     = (tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y);
	  output[id]    = sqrtf(d2);											//save the Euclidean DT
    }
}


__global__ void kernelSkel(float* output, bool* fg, short xm, short ym, 
						   short xM, short yM, short size, float threshold, float length)	
																			//Input:    pbaTexColor: closest-site-ids per pixel
																			//			pbaTexParam: labels for sites (only valid at site locations)
{																			//Output:	output: binary thresholded skeleton
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (tx>xm && ty>ym && tx<xM && ty<yM) 
	{
  	  int    id     = TOID(tx, ty, size);
	  int    Id     = id;
	  short2 voroid = tex1Dfetch(pbaTexColor,id);							//get the closest-site to tx,ty into voroid.x,.y
	  int    id2    = TOID(voroid.x,voroid.y,size);							//convert the site's coord to an index into pbaTexParam[], the site-label-texture
	  float  imp    = tex1Dfetch(pbaTexParam,id2);							//get the site's label
	  float  d2     = (tx-voroid.x)*(tx-voroid.x)+(ty-voroid.y)*(ty-voroid.y);
	  float  dt     = sqrtf(d2);											//save the Euclidean DT

	         ++id;															//TOID(tx+1,ty,size)
	         voroid = tex1Dfetch(pbaTexColor,id);							//
	         id2    = TOID(voroid.x,voroid.y,size);							//
	  float  imp_r  = tex1Dfetch(pbaTexParam,id2);							//

	         id     += size-1;												//TOID(tx,ty+1,size)
	         voroid = tex1Dfetch(pbaTexColor,id);							//
	         id2    = TOID(voroid.x,voroid.y,size);							//
	  float  imp_u  = tex1Dfetch(pbaTexParam,id2);							//

      float imp_dx  = fabsf(imp_r-imp);
	  float imp_dy  = fabsf(imp_u-imp);
	  float Imp     = max(imp_dx,imp_dy);

	  // importance naar salience?
	  Imp = min(Imp,fabsf(length-Imp));    //importance
      float saliency = Imp / dt;
	  //if (fg[id] && Imp>=threshold) output[Id] = 1;	//wang.The original one is fg[id].				//By filling only 1-values, we reduce memory access somehow (writing to output[] is expensive)
	  //if (fg[id] && saliency>=threshold) output[Id] = saliency;
	  if (fg[id] && Imp > 3.0 && saliency>=threshold) output[Id] = saliency;	//wang.The original one is fg[id].				//By filling only 1-values, we reduce memory access somehow (writing to output[] is expensive)
		
	} 
	
	//WARNING: this kernel may sometimes creates 2-pixel-thick branches.. Study the AFMM original code to see if this is correct.
}
 


#define X 1

__constant__ const															//REMARK: put following constants (for kernelTopology) in CUDA constant-memory, as this gives a huge speed difference
unsigned char topo_patterns[][9] =		{ {0,0,0,							//These are the 3x3 templates that we use to detect skeleton endpoints
										   0,X,0,							//(with four 90-degree rotations for each)
										   0,X,0},
										  {0,0,0,
										   0,X,0,
										   0,0,X},
										  {0,0,0,
										   0,X,0,
										   0,X,X},
										  {0,0,0,
										   0,X,0,
										   X,X,0} 
										};
										
#define topo_NPATTERNS  4														//Number of patterns we try to match (for kernelTopology)
																				//REMARK: #define faster than __constant__

__constant__ const unsigned char topo_rot[][9] = { {0,1,2,3,4,5,6,7,8}, {2,5,8,1,4,7,0,3,6}, {8,7,6,5,4,3,2,1,0}, {6,3,0,7,4,1,8,5,2} };
																				//These encode the four 90-degree rotations of the patterns (for kernelTopology);

__device__ unsigned int topo_gc			= 0;
__device__ unsigned int topo_gc_last	= 0;


__global__ void kernelTopology(unsigned char* output, short2* output_set, short xm, short ym, short xM, short yM, short size, int maxpts)	
{
	const int tx = blockIdx.x * blockDim.x + threadIdx.x;
	const int ty = blockIdx.y * blockDim.y + threadIdx.y;
		
	unsigned char t[9];
	
	if (tx>xm && ty>ym && tx<xM-1 && ty<yM-1)									//careful not to index outside the image; take into account the template size too
	{	
	   int    id = TOID(tx, ty, size);	 
	   unsigned char  p  = tex1Dfetch(pbaTexGray,id);							//get the skeleton pixel at tx,ty
	   if (p)																	//if the pixel isn't skeleton, nothing to do
	   {
	     unsigned char idx=0;
		 for(int j=ty-1;j<=ty+1;++j)											//read the template into t[] for easier use
		 {
		   int id = TOID(tx-1, j, size);
	       for(int i=0;i<=2;++i,++id,++idx)
		      t[idx] = tex1Dfetch(pbaTexGray,id);								//get the 3x3 template centered at the skel point tx,ty
		 }
		  
		 for(unsigned char r=0;r<4;++r)											//try to match all rotations of a pattern:
		 {
		   const unsigned char* rr = topo_rot[r];
	       for(unsigned char p=0;p<topo_NPATTERNS;++p)							//try to match all patterns:
	       {
	         const unsigned char* pat = topo_patterns[p];
			 unsigned char j = (p==0)? 0 : 7;									//Speedup: for all patterns except 1st, check only last 3 entries, the first 6 are identical for all patterns
			 for(;j<9;++j)														//try to match rotated pattern vs actual pattern
			   if (pat[j]!=t[rr[j]]) break;										//this rotation failed
			 if (j<6) break;													//Speedup: if we have a mismatch on the 1st 6 pattern entries, then none of the patterns can match
																				//		   since all templates have the same first 6 entries.

			 if (j==9)															//this rotation succeeded: mark the pixel as a topology event and we're done
			 {	
				int crt_gc = atomicInc(&topo_gc,maxpts);						//REMARK: this serializes (compacts) all detected endpoints in one array.			
				output_set[crt_gc] = make_short2(tx,ty);						//To do this, we use an atomic read-increment-return on a global counter, 
																				//which is guaranteed to give all threads unique consecutive indexes in the array.
			    output[id] = 1;													//Also create the topology image
				return;
			 }
		   }
		 }
	   }
	}
	else																		//Last thread: add zero-marker to the output point-set, so the reader knows how many points are really in there
	if (tx==xM-1 && ty==yM-1)													//Also reset the global vector counter topo_gc, for the next parallel-run of this function
	{ topo_gc_last = topo_gc; topo_gc = 0; }									//We do this in the last thread so that no one modifies topo_gc from now on. 
																				//REMARK: this seems to be the only way I can read a __device__ variable back to the CPU
}




void skelft2DParams(int floodBand_, int maurerBand_, int colorBand_)		//Set up some params of the FT algorithm	
{
  floodBand   = floodBand_;
  maurerBand  = maurerBand_;
  colorBand   = colorBand_;
}





// Compute 2D FT / Voronoi diagram of a set of sites
// siteParam:   Site parameterization. 0 = non-site points; >0 = site parameter value.
// output:		FT. The (x,y) at (i,j) are the coords of the closest site to (i,j)
// size:        Texture size (pow 2)
void skelft2DFT(short* output, float* siteParam, short xm, short ym, short xM, short yM, int size) 
{
    skelft2DInitializeInput(siteParam,size);								    // Initialization of already-allocated data structures

    skel2DFTCompute(xm, ym, xM, yM, floodBand, maurerBand, colorBand);			// Compute FT
	
    // Copy FT to CPU, if required
    if (output) hipMemcpy(output, pbaTextures[1], size*size*sizeof(short2), hipMemcpyDeviceToHost);
}








void skelft2DDT(short* outputDT, float threshold,								//Compute (thresholded) DT (into pbaTextures[2]) from resident FT (in pbaTextures[1])	
					  short xm, short ym, short xM, short yM)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    hipBindTexture(0, pbaTexColor, pbaTextures[1]);							//Used to read the FT from

	if (threshold>=0)
	{
	  xm -= threshold; if (xm<0) xm=0;
	  ym -= threshold; if (ym<0) ym=0;
	  xM += threshold; if (xM>pbaTexSize-1) xM=pbaTexSize-1;
	  yM += threshold; if (yM>pbaTexSize-1) yM=pbaTexSize-1;
	
      kernelThresholdDT<<< grid, block >>>((unsigned char*)pbaTextures[2], pbaTexSize, threshold*threshold, xm-1, ym-1, xM+1, yM+1);    
      hipUnbindTexture(pbaTexColor);
	
	  //Copy thresholded image to CPU
	  if (outputDT) hipMemcpy(outputDT, (unsigned char*)pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	}
	else
	{
	  xm = ym = 0; xM = yM = pbaTexSize-1;
	  kernelDT <<< grid, block >>>((short*)pbaTextures[2], pbaTexSize, threshold*threshold, xm-1, ym-1, xM+1, yM+1);
      hipUnbindTexture(pbaTexColor);
	  //Copy thresholded image to CPU
	  if (outputDT) hipMemcpy(outputDT, pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(short), hipMemcpyDeviceToHost);
	}
}




void skelft2DSkeleton(float* outputSkel, bool* fg, float length, float threshold,	//Compute thresholded skeleton (into pbaTextures[3]) from resident FT (in pbaTextures[1])
					  short xm,short ym,short xM,short yM)						
{																				//length:     boundary length
	dim3 block = dim3(BLOCKX,BLOCKY);											//threshold:  skeleton importance min-value (below this, we ignore branches)
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);
	
    hipBindTexture(0, pbaTexColor, pbaTextures[1]);							//Used to read the resident FT
	hipBindTexture(0, pbaTexParam, pbaTexSiteParam);							//Used to read the resident boundary parameterization	
	hipMemset(pbaTextures[3],0,sizeof(float)*pbaTexSize*pbaTexSize);	//Faster to zero result and then fill only 1-values (see kernel)
	
    kernelSkel<<< grid, block >>>((float*)pbaTextures[3], fg, xm, ym, xM-1, yM-1, pbaTexSize, threshold, length);
	
    hipUnbindTexture(pbaTexColor);
	hipUnbindTexture(pbaTexParam);
	
	//Copy skeleton to CPU
	if (outputSkel) hipMemcpy(outputSkel, pbaTextures[3], pbaTexSize * pbaTexSize * sizeof(float), hipMemcpyDeviceToHost);
}




void skelft2DTopology(unsigned char* outputTopo, int* npts, short* outputPoints, //Compute topology-points of the resident skeleton (in pbaTextures[3])
					  short xm,short ym,short xM,short yM)					
{
    int maxpts = (npts)? *npts : pbaTexSize*pbaTexSize;							//This is the max # topo-points we are going to return in outputPoints[]

	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);
	
    hipBindTexture(0, pbaTexGray, pbaTextures[3]);								//Used to read the resident skeleton
	hipMemset(pbaTextures[4],0,sizeof(unsigned char)*pbaTexSize*pbaTexSize);	//Faster to zero result and then fill only 1-values (see kernel)

    unsigned int zero = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(topo_gc),&zero,sizeof(unsigned int),0,hipMemcpyHostToDevice);		//Set topo_gc to 0

    kernelTopology<<< grid, block >>>((unsigned char*)pbaTextures[4], pbaTextures[5], xm, ym, xM, yM, pbaTexSize, maxpts+1);
	
    hipUnbindTexture(pbaTexGray);

	if (outputPoints && maxpts)													//If output-point vector desired, copy the end-points, put in pbaTexture[5] as a vector of short2's, 
	{																			//into caller space. We copy only 'maxpts' elements, as the user instructed us.
	    unsigned int num_pts;
		hipMemcpyFromSymbol(&num_pts,HIP_SYMBOL(topo_gc_last),sizeof(unsigned int),0,hipMemcpyDeviceToHost);		//Get #topo-points we have detected from the device-var from CUDA
		if (npts && num_pts)																			//Copy the topo-points to caller		
		   hipMemcpy(outputPoints,pbaTextures[5],num_pts*sizeof(short2),hipMemcpyDeviceToHost);
		if (npts) *npts = num_pts;												//Return #detected topo-points to caller								
	}
		
	if (outputTopo)																//If topology image desired, copy it into user space
		hipMemcpy(outputTopo,pbaTextures[4],pbaTexSize*pbaTexSize*sizeof(unsigned char), hipMemcpyDeviceToHost);
}




__global__ void kernelSiteFromSkeleton(short2* outputSites, int size)						//Initialize the Voronoi textures from the sites' encoding texture (parameterization)
{																							//REMARK: we interpret 'inputVoro' as a 2D texture, as it's much easier/faster like this
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx<size && ty<size)																	//Careful not to go outside the image..
	{
	  int i = TOID(tx,ty,size);
	  unsigned char param = tex1Dfetch(pbaTexGray,i);										//The sites-param has non-zero (parameter) values precisely on non-boundary points

	  short2& v = outputSites[i];
	  v.x = v.y = MARKER;																	//Non-boundary points are marked as 0 in the parameterization. Here we will compute the FT.
	  if (param)																			//These are points which define the 'sites' to compute the FT/skeleton (thus, have FT==identity)
	  {																						//We could use an if-then-else here, but it's faster with an if-then
	     v.x = tx; v.y = ty;
	  }
	}
}




__global__ void kernelSkelInterpolate(float* output, int size)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx<size && ty<size)																	//Careful not to go outside the image..
	{
  	  int    id     = TOID(tx, ty, size);
	  short2 vid    = tex1Dfetch(pbaTexColor,id);							
	  float  T      = sqrtf((tx-vid.x)*(tx-vid.x)+(ty-vid.y)*(ty-vid.y));
	  short2 vid2   = tex1Dfetch(pbaTexColor2,id);							
	  float  D      = sqrtf((tx-vid2.x)*(tx-vid2.x)+(ty-vid2.y)*(ty-vid2.y));
	  float  B      = ((D)? min(T/2/D,0.5f):0.5) + 0.5*((T)? max(1-D/T,0.0f):0);
	  output[id]    = B;
	}
}




void skel2DSkeletonDT(float* outputSkelDT,short xm,short ym,short xM,short yM)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    hipBindTexture(0,pbaTexGray,pbaTextures[3]);							//Used to read the resident binary skeleton
    kernelSiteFromSkeleton<<<grid,block>>>(pbaTextures[0],pbaTexSize);		//1. Init pbaTextures[0] with sites on skeleton i.e. from pbaTexGray
	hipUnbindTexture(pbaTexGray);
		
	//!!Must first save pbaTextures[1] since we may need it later..
	hipMemcpy(pbaTextures[5],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    skel2DFTCompute(xm, ym, xM, yM, floodBand, maurerBand, colorBand);		//2. Compute FT of the skeleton into pbaTextures[6]
    hipMemcpy(pbaTextures[6],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    hipMemcpy(pbaTextures[1],pbaTextures[5],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    
	//Compute interpolation		
    hipBindTexture(0,pbaTexColor,pbaTextures[1]);							// FT of boundary
    hipBindTexture(0,pbaTexColor2,pbaTextures[6]);							// FT of skeleton
	kernelSkelInterpolate<<<grid,block>>>((float*)pbaTextures[0],pbaTexSize);
	hipUnbindTexture(pbaTexColor);
	hipUnbindTexture(pbaTexColor2);
	if (outputSkelDT) hipMemcpy(outputSkelDT, pbaTextures[0], pbaTexSize * pbaTexSize * sizeof(float), hipMemcpyDeviceToHost);
}

void skel2DSkeletonFT(short* outputSkelFT,short xm,short ym,short xM,short yM)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    hipBindTexture(0,pbaTexGray,pbaTextures[3]);							//Used to read the resident binary skeleton
    kernelSiteFromSkeleton<<<grid,block>>>(pbaTextures[0],pbaTexSize);		//1. Init pbaTextures[0] with sites on skeleton i.e. from pbaTexGray
	hipUnbindTexture(pbaTexGray);
		
	//!!Must first save pbaTextures[1] since we may need it later..
	hipMemcpy(pbaTextures[5],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);
    skel2DFTCompute(xm, ym, xM, yM, floodBand, maurerBand, colorBand);		//2. Compute FT of the skeleton into pbaTextures[6]
    hipMemcpy(pbaTextures[6],pbaTextures[1],pbaTexSize*pbaTexSize*sizeof(short2),hipMemcpyDeviceToDevice);

	if (outputSkelFT) hipMemcpy(outputSkelFT, pbaTextures[6], pbaTexSize * pbaTexSize * sizeof(short2), hipMemcpyDeviceToHost);
}



__device__  bool fill_gc;														//Indicates if a fill-sweep did fill anything or not


__global__ void kernelFill(unsigned char* output, int size, unsigned char bg, unsigned char fg, short xm, short ym, short xM, short yM, bool ne)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
		
	if (tx>xm && ty>ym && tx<xM && ty<yM)										//careful not to index outside the image..
	{	
	  int    id0 = TOID(tx, ty, size);
	  unsigned char val = tex1Dfetch(pbaTexGray,id0);							//
	  if (val==fg)																//do we have a filled pixel? Then fill all to left/top/up/bottom of it which is background
	  {
	    bool fill = false;
		int id = id0;
		if (ne)																	//fill in north-east direction:
		{
			for(short x=tx+1;x<xM;++x)											//REMARK: here and below, the interesting thing is that it's faster, by about 10-15%, to fill a whole
			{																	//        scanline rather than oly until the current block's borders (+1). The reason is that filling a whole
																				//		  scanline decreases the total #sweeps, which seems to be the limiting speed factor
			  if (tex1Dfetch(pbaTexGray,++id)!=bg) break;
			  output[id] = fg; fill = true;
			}

			id = id0;
			for(short y=ty-1;y>ym;--y)
			{
			  if (tex1Dfetch(pbaTexGray,id-=size)!=bg) break;
			  output[id] = fg; fill = true;
			}
		}
		else																	//fill in south-west direction:
		{
			for(short x=tx-1;x>xm;--x)
			{
			  if (tex1Dfetch(pbaTexGray,--id)!=bg) break;
			  output[id] = fg; fill = true;
			}

			id = id0;
			for(short y=ty+1;y<yM;++y)
			{
			  if (tex1Dfetch(pbaTexGray,id+=size)!=bg) break;
			  output[id] = fg; fill = true;
			}
		}
		
	    if (fill) fill_gc = true;												//if we filled anything, inform caller; we 'gather' this info from a local var into the
																				//global var here, since it's faster than writing the global var in the for loops
	  }	  
    }
}




__global__ void kernelFillHoles(unsigned char* output, int size, unsigned char bg, unsigned char fg, unsigned char fill_fg)
{
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (tx>=0 && ty>=0 && tx<size && ty<size)									//careful not to index outside the image..
	{	
  	  int            id = TOID(tx, ty, size);
	  unsigned char val = tex1Dfetch(pbaTexGray,id);							//
	  if (val==fill_fg)
	     output[id] = bg;
	  else if (val==bg)
	     output[id] = fg;	 
	}
}


int skelft2DFill(unsigned char* outputFill, short sx, short sy, short xm, short ym, short xM, short yM, unsigned char fill_value)
{
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);

    unsigned char background;
	int id = sy * pbaTexSize + sx;
	hipMemcpy(&background,(unsigned char*)pbaTextures[2]+id,sizeof(unsigned char),hipMemcpyDeviceToHost); //See which is the value we have to fill from (sx,sy)
	
	hipMemset(((unsigned char*)pbaTextures[2])+id,fill_value,sizeof(unsigned char));					//Fill the seed (x,y) on the GPU	

	hipBindTexture(0, pbaTexGray, pbaTextures[2]);														//Used to read the thresholded DT

	int iter=0;
	bool xy = true;																						//Direction of filling for current sweep: either north-east or south-west
																										//This kind of balances the memory-accesses nicely over kernel calls
	for(;;++iter,xy=!xy)																				//Keep filling a sweep at a time until we have no background pixels anymore
	{	
	   bool filled = false;																				//Initialize flag: we didn't fill anything in this sweep
	   hipMemcpyToSymbol(HIP_SYMBOL(fill_gc),&filled,sizeof(bool),0,hipMemcpyHostToDevice);						//Pass flag to CUDA
       kernelFill<<<grid, block>>>((unsigned char*)pbaTextures[2],pbaTexSize,background,fill_value,xm,ym,xM,yM,xy);	
																										//One fill sweep	   
	   hipMemcpyFromSymbol(&filled,HIP_SYMBOL(fill_gc),sizeof(bool),0,hipMemcpyDeviceToHost);						//See if we filled anything in this sweep
	   if (!filled) break;																				//Nothing filled? Then we're done, the image didn't change
	}
	hipUnbindTexture(pbaTexGray);
		
	if (outputFill) hipMemcpy(outputFill, (unsigned char*)pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	return iter;																						//Return #iterations done for the fill - useful as a performance measure for caller
}



int skelft2DFillHoles(unsigned char* outputFill, short sx, short sy, unsigned char foreground)
{
    unsigned char background;
	unsigned char fill_value = 128;
	int id = sy * pbaTexSize + sx;
	hipMemcpy(&background,(unsigned char*)pbaTextures[2]+id,sizeof(unsigned char),hipMemcpyDeviceToHost); //See which is the value at (sx,sy)

	int iter = skelft2DFill(0,sx,sy,0,0,pbaTexSize,pbaTexSize,fill_value);								//First, fill the background surrounding the image with some special value
		
	dim3 block = dim3(BLOCKX,BLOCKY);
	dim3 grid  = dim3(pbaTexSize/block.x,pbaTexSize/block.y);
	

    hipBindTexture(0, pbaTexGray, pbaTextures[2]);														//Used to read the thresholded DT

    kernelFillHoles<<<grid, block>>>((unsigned char*)pbaTextures[2],pbaTexSize,background,foreground,fill_value);

    hipUnbindTexture(pbaTexGray);
	
	if (outputFill) hipMemcpy(outputFill, (unsigned char*)pbaTextures[2], pbaTexSize * pbaTexSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	return iter;
}

